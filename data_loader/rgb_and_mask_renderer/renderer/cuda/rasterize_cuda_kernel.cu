#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <tuple>


__global__ void forward_face_index_map_cuda_kernel(
        const float* faces,
        const int batch_size,
        const int num_faces,
        const int image_height,
        const int image_width,
        const float near,
        const float far,
        int32_t* face_index_map,
        float* weight_map,
        float* depth_map,
        int32_t* lock_map) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * num_faces) {
        return;
    }
    const int ih = image_height;
    const int iw = image_width;
    const int bn = i / num_faces;
    const int fn = i % num_faces;

    const float* face = &faces[i * 9];

    /* pi[0], pi[1], pi[2] = leftmost, middle, rightmost points */
    int pi[3];
    if (face[0] < face[3]) {
        if (face[6] < face[0]) pi[0] = 2; else pi[0] = 0;
        if (face[3] < face[6]) pi[2] = 2; else pi[2] = 1;
    } else {
        if (face[6] < face[3]) pi[0] = 2; else pi[0] = 1;
        if (face[0] < face[6]) pi[2] = 2; else pi[2] = 0;
    }
    for (int k = 0; k < 3; k++) {
      if (pi[0] != k && pi[2] != k) {
          pi[1] = k;
      }
    }

    /* p[num][xyz]: x, y is normalized from [-1, 1] to [0, 3 * ih or iw - 1]. */
    float p[3][3];
    for (int num = 0; num < 3; num++) {
        for (int dim = 0; dim < 3; dim++) {
            if (dim == 0) {
                p[num][dim] = iw + 0.5 * (face[3 * pi[num] + dim] * iw + iw - 1);
            } else if (dim == 1) {
                p[num][dim] = ih + 0.5 * (face[3 * pi[num] + dim] * ih + ih - 1);
            } else {
                p[num][dim] = face[3 * pi[num] + dim];
            }
        }
    }
    if (p[0][0] == p[2][0]) return; // line, not triangle

    /* compute face_inv */
    float face_inv[9] = {
        p[1][1] - p[2][1], p[2][0] - p[1][0], p[1][0] * p[2][1] - p[2][0] * p[1][1],
        p[2][1] - p[0][1], p[0][0] - p[2][0], p[2][0] * p[0][1] - p[0][0] * p[2][1],
        p[0][1] - p[1][1], p[1][0] - p[0][0], p[0][0] * p[1][1] - p[1][0] * p[0][1]};

    float face_inv_denominator = (
        p[2][0] * (p[0][1] - p[1][1]) +
        p[0][0] * (p[1][1] - p[2][1]) +
        p[1][0] * (p[2][1] - p[0][1]));

    for (int k = 0; k < 9; k++) {
        face_inv[k] /= face_inv_denominator;
    }

    const int xi_min = max(ceil(p[0][0]), 0.0);
    const int xi_max = min(p[2][0], 3 * iw - 1.0);
    for (int xi = xi_min; xi <= xi_max; xi++) {
        /* compute yi_min and yi_max */
        float yi1, yi2;
        if (xi <= p[1][0]) {
            if (p[1][0] - p[0][0] != 0) {
                yi1 = (p[1][1] - p[0][1]) / (p[1][0] - p[0][0]) * (xi - p[0][0]) + p[0][1];
            } else {
                yi1 = p[1][1];
            }
        } else {
            if (p[2][0] - p[1][0] != 0) {
                yi1 = (p[2][1] - p[1][1]) / (p[2][0] - p[1][0]) * (xi - p[1][0]) + p[1][1];
            } else {
                yi1 = p[1][1];
            }
        }
        yi2 = (p[2][1] - p[0][1]) / (p[2][0] - p[0][0]) * (xi - p[0][0]) + p[0][1];

        const int yi_min = max(ceil(min(yi1, yi2)), 0.0);
        const int yi_max = min(max(yi1, yi2), 3 * ih - 1.0);

        for (int yi = yi_min; yi <= yi_max; yi++) {

            int index = bn * 3 * ih * 3 * iw + yi * 3 * iw + xi;
            /* compute w = face_inv * p */
            float w[3];
            for (int k = 0; k < 3; k++) {
                w[k] = face_inv[3 * k + 0] * xi + face_inv[3 * k + 1] * yi + face_inv[3 * k + 2];
            }
            /* sum(w) -> 1, 0 < w < 1 */
            float w_sum = 0;
            for (int k = 0; k < 3; k++) {
                w[k] = min(max(w[k], 0.0), 1.0);
                w_sum += w[k];
            }
            for (int k = 0; k < 3; k++) w[k] /= w_sum;
            /* compute 1 / zp = sum(w / z) */
            const float zp = 1.0 / (w[0] / p[0][2] + w[1] / p[1][2] + w[2] / p[2][2]);
            if (zp <= near || far <= zp) continue;

            /* lock and update */
            bool locked = false;
            do {
                if (locked = atomicCAS(&lock_map[index], 0, 1) == 0) {
                    if (zp < atomicAdd(&depth_map[index], 0)) {
                        float record = 0;
                        atomicExch(&depth_map[index], zp);
                        atomicExch(&face_index_map[index], fn);
                        for (int k = 0; k < 3; k++) {
                            atomicExch(&weight_map[3 * index + pi[k]], w[k]);
                        }
                        record += atomicAdd(&depth_map[index], 0.);
                        record += atomicAdd(&face_index_map[index], 0.);
                        if (record > 0) atomicExch(&lock_map[index], 0);
                    } else {
                        atomicExch(&lock_map[index], 0);
                    }
                }
            } while (!locked);

        }
    }
}

__global__ void forward_texture_sampling_cuda_kernel(
		const float* faces,
		const float* textures,
	  const int32_t* face_index_map,
	  const float* weight_map,
    const size_t batch_size,
    const int num_faces,
    const int image_height,
    const int image_width,
    const int texture_size,
		float* rgb_map,
		int32_t* mask_map) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * 3 * image_height * 3 * image_width) {
        return;
    }

    const int ts = texture_size;
    const int face_index = face_index_map[i];
    float* pixel = &rgb_map[i * ts];
    int32_t* pixel_mask = &mask_map[i];
    
    if (face_index >= 0) {
        /*
            from global variables:
            batch number, num of faces, image_size, face[v012][RGB], pixel[RGB], weight[v012],
            texture[ts][RGB];
        */
        const int bn = i / (3 * image_height * 3 * image_width);
        const int nf = num_faces;

        const float* texture = &textures[(bn * nf + face_index) * ts * 3];
        const float* weight = &weight_map[i * 3];
    
        /* blend */
        for (int k = 0; k < ts; k++) {
            for (int j = 0; j < 3; j++) {
                pixel[k] += weight[j] * texture[ts * j + k];
            }
        }
        mask_map[i] = 1;
    }
}


std::tuple<at::Tensor, at::Tensor, at::Tensor> forward_cuda(
        const at::Tensor& faces,
        const at::Tensor& textures,
        const int image_height,
        const int image_width,
        const float near,
        const float far) {

    const int batch_size = faces.size(0);
    const int num_faces = faces.size(1);
    const int texture_size = 3;
    const int threads = 512;

    auto int_opts = textures.options().dtype(at::kInt);
    auto float_opts = textures.options().dtype(at::kFloat);

    at::Tensor face_index_map = at::full({batch_size, 3 * image_height, 3 * image_width}, -1, int_opts);
    at::Tensor weight_map = at::empty({batch_size, 3 * image_height, 3 * image_width, 3}, float_opts);
    at::Tensor depth_map = at::full({batch_size, 3 * image_height, 3 * image_width}, far, float_opts);
    at::Tensor lock_map = at::full({batch_size, 3 * image_height, 3 * image_width}, 0, int_opts);

    const dim3 blocks1 ((batch_size * num_faces - 1) / threads +1);

    forward_face_index_map_cuda_kernel<<<blocks1, threads>>>(
        faces.data_ptr<float>(),
        batch_size,
        num_faces,
        image_height,
        image_width,
        near,
        far,
        face_index_map.data_ptr<int32_t>(),
        weight_map.data_ptr<float>(),
        depth_map.data_ptr<float>(),
        lock_map.data_ptr<int32_t>());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)  {
        printf("Error in forward_face_index_map: %s\n", hipGetErrorString(err));
    }

    at::Tensor rgb_map = at::full({batch_size, 3 * image_height, 3 * image_width, texture_size}, 0.0f, float_opts);
    at::Tensor mask_map = at::full({batch_size, 3 * image_height, 3 * image_width}, 0, int_opts);

    const dim3 blocks2 ((batch_size * 3 * image_height * 3 * image_width - 1) / threads + 1);

    forward_texture_sampling_cuda_kernel<<<blocks2, threads>>>(
        faces.data_ptr<float>(),
        textures.data_ptr<float>(),
        face_index_map.data_ptr<int32_t>(),
        weight_map.data_ptr<float>(),
        batch_size,
        num_faces,
        image_height,
        image_width,
        texture_size,
        rgb_map.data_ptr<float>(),
        mask_map.data_ptr<int32_t>());

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in forward_texture_sampling: %s\n", hipGetErrorString(err));
    }

    return std::make_tuple(rgb_map, mask_map, depth_map);
}
